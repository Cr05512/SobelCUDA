#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <thread>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <time.h>
#include <chrono>
#include <omp.h>
#include <sched.h>


/*
Compiling: Requires a Nvidia CUDA capable graphics card and the Nvidia GPU Computing Toolkit.
 *            Linux: nvcc -Wno-deprecated-gpu-targets -O3 -o test main.cu -Xcompiler -fopenmp `pkg-config --cflags --libs opencv`
 */

#define GRIDVAL 16.0
#define meanLength 60
typedef unsigned char byte;

void sobel_cpu(const cv::Mat* orig_gs, cv::Mat* edges_cpu, const unsigned int width, const unsigned int height);
void sobel_omp(const cv::Mat* orig_gs, cv::Mat* edges_omp, const unsigned int width, const unsigned int height);
int avg(int* fpsMeanVec);

__global__ void sobel_gpu(const byte* orig, byte* gpu, const unsigned int width, const unsigned int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
             (    orig[(y-1)*width + (x+1)]) + ( 2*orig[y*width+(x+1)]) + (   orig[(y+1)*width+(x+1)]);
        dy = (    orig[(y-1)*width + (x-1)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+1)]) +
             (-1* orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
        gpu[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
    }
}

std::string gstreamer_pipeline (int capture_width, int capture_height, int display_width, int display_height, int framerate, int flip_method) {
    return "nvarguscamerasrc ! video/x-raw(memory:NVMM), width=(int)" + std::to_string(capture_width) + ", height=(int)" +
           std::to_string(capture_height) + ", format=(string)NV12, framerate=(fraction)" + std::to_string(framerate) +
           "/1 ! nvvidconv flip-method=" + std::to_string(flip_method) + " ! video/x-raw, width=(int)" + std::to_string(display_width) + ", height=(int)" +
           std::to_string(display_height) + ", format=(string)BGRx ! videoconvert ! video/x-raw, format=(string)BGR ! appsink";
}

int main (int argc, char* argv[])
{
    try
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, 0);
        int cores = devProp.multiProcessorCount;
        switch (devProp.major)
        {
        case 2: // Fermi
            if (devProp.minor == 1) cores *= 48;
            else cores *= 32; break;
        case 3: // Kepler
            cores *= 192; break;
        case 5: // Maxwell
            cores *= 128; break;
        case 6: // Pascal
            if (devProp.minor == 1) cores *= 128;
            else if (devProp.minor == 0) cores *= 64;
            break;
        }
        time_t rawTime;time(&rawTime);
        struct tm* curTime = localtime(&rawTime);
        char timeBuffer[80] = "";
        strftime(timeBuffer, 80, "Sobel Edge Detector Benchamrks (%c)\n", curTime);
        printf("%s", timeBuffer);
        printf("CPU: %d hardware threads\n", std::thread::hardware_concurrency());
        printf("GPGPU: %s, CUDA %d.%d, %zd Mbytes global memory, %d CUDA cores\n",
        devProp.name, devProp.major, devProp.minor, devProp.totalGlobalMem / 1048576, cores);

        

        cv::namedWindow("Sobel Edge Detector",cv::WINDOW_AUTOSIZE);
        unsigned int width = 640;
        unsigned int height = 480;
        // unsigned int framerate = 120;
        // unsigned int flip_method = 0;
        // std::string pipeline = gstreamer_pipeline(width,
        //     height,
        //     width,
        //     height,
        //     framerate,
        //     flip_method);
        
        //cv::VideoCapture camera(pipeline, cv::CAP_GSTREAMER);
        cv::VideoCapture camera(0);
        if(!camera.isOpened())
            return -1;
        //cv::resizeWindow("Sobel Edge Detector", frameWidth, frameHeight);
        //camera.set(cv::CAP_PROP_FRAME_WIDTH, width);
        //camera.set(cv::CAP_PROP_FRAME_HEIGHT, height);
        cv::Mat* orig, *orig_gs, *edges;
        //unsigned int width, height = 0;
        //width = camera.get(cv::CAP_PROP_FRAME_WIDTH);
        //height = camera.get(cv::CAP_PROP_FRAME_HEIGHT);
        
        orig = new cv::Mat(height,width,CV_8UC3);
        orig_gs = new cv::Mat(height,width,CV_8UC1);
        edges = new cv::Mat(height,width,CV_8UC1);
        byte *gpu_orig, *gpu_sobel;
        hipMalloc((void**)&gpu_orig,(width*height*sizeof(byte)));
        hipMalloc((void**)&gpu_sobel,(width*height*sizeof(byte)));

        dim3 threadsPerBlock(GRIDVAL, GRIDVAL, 1);
        dim3 numBlocks(ceil(width/GRIDVAL), ceil(height/GRIDVAL), 1);
        //std::cout << ceil(width/GRIDVAL)*ceil(height/GRIDVAL) << std::endl;

        uint8_t key = 0;
        int8_t tmp = 0;
        auto c = std::chrono::system_clock::now();
        std::chrono::duration<double> time;

        std::ostringstream buf;
        int* fpsMeanVec = new int[meanLength];
        memset(fpsMeanVec,0,meanLength*sizeof(int));
        uint8_t counter = 0;

        for(;;){
            camera >> *orig;
            cv::cvtColor(*orig, *orig_gs, CV_BGR2GRAY, 0);
            tmp = cv::waitKey(1);
            if(tmp != -1){
                key = tmp;
            }
            switch(key){
                case 99:
                    c = std::chrono::system_clock::now();
                    sobel_cpu(orig_gs, edges, width, height);
                    time = std::chrono::system_clock::now() - c;
                    fpsMeanVec[counter] = (int)(1/time.count());
                    counter++;
                    buf << "Mode: CPU" << "," << "  FPS: " << avg(fpsMeanVec);
                    putText(*edges, buf.str(), cv::Point(10, 30), cv::FONT_HERSHEY_PLAIN, 2.0, cv::Scalar(255, 255, 255), 1, cv::LINE_AA);
                    cv::imshow("Sobel Edge Detector", *edges);
                    buf.str("");
                    buf.clear();
                    break;
                case 111: 
                    c = std::chrono::system_clock::now();
                    sobel_omp(orig_gs, edges, width, height);
                    time = std::chrono::system_clock::now() - c;
                    fpsMeanVec[counter] = (int)(1/time.count());
                    counter++;
                    buf << "Mode: OMP" << "," << "  FPS: " << avg(fpsMeanVec);
                    putText(*edges, buf.str(), cv::Point(10, 30), cv::FONT_HERSHEY_PLAIN, 2.0, cv::Scalar(255, 255, 255), 1, cv::LINE_AA);
                    cv::imshow("Sobel Edge Detector", *edges);
                    buf.str("");
                    buf.clear();
                    break;
                case 103:
                    c = std::chrono::system_clock::now();
                    hipMemcpy(gpu_orig, orig_gs->data, (width*height*sizeof(byte)), hipMemcpyHostToDevice);
                    //hipMemset(gpu_sobel, 0, (width*height*sizeof(byte)));
                    sobel_gpu<<<numBlocks, threadsPerBlock>>>(gpu_orig, gpu_sobel, width, height);
                    hipDeviceSynchronize(); // waits for completion, returns error code
                    hipMemcpy(edges->data, gpu_sobel, (width*height), hipMemcpyDeviceToHost);
                    time = std::chrono::system_clock::now() - c;
                    fpsMeanVec[counter] = (int)(1/time.count());
                    counter++;
                    buf << "Mode: GPU" << "," << "  FPS: " << avg(fpsMeanVec);
                    putText(*edges, buf.str(), cv::Point(10, 30), cv::FONT_HERSHEY_PLAIN, 2.0, cv::Scalar(255, 255, 255), 1, cv::LINE_AA);
                    cv::imshow("Sobel Edge Detector", *edges);
                    buf.str("");
                    buf.clear();
                    break;
                case 27:
                    delete orig; delete orig_gs; delete edges;
                    hipFree(gpu_orig); hipFree(gpu_sobel);
                    camera.release();
                    cv::destroyAllWindows();
                    return 0;
                default:
                    cv::imshow("Sobel Edge Detector", *orig);
                    break;
            }
            if(counter==meanLength-1){
                counter = 0;
            }
            //std::cout << "FPS: " << (int)(1/time.count()) << std::endl;
            
        }

    }
    catch(const cv::Exception& ex)
    {
        std::cout << "Error: " << ex.what() << std::endl;
    }
    return 0;
}

void sobel_cpu(const cv::Mat* orig_gs, cv::Mat* edges_cpu, const unsigned int width, const unsigned int height) {
    omp_set_num_threads(1);
    
    #pragma omp parallel for
    for(int y = 1; y < height-1; y++) {
        for(int x = 1; x < width-1; x++) {
            int dx = (-1* (orig_gs->at<uint8_t>(y-1,x-1))) + (-2*(orig_gs->at<uint8_t>(y,x-1))) + (-1*(orig_gs->at<uint8_t>(y+1,x-1))) +
            (orig_gs->at<uint8_t>(y-1,x+1)) + (2*(orig_gs->at<uint8_t>(y,x+1))) + ((orig_gs->at<uint8_t>(y+1,x+1)));
            int dy = (orig_gs->at<uint8_t>(y-1,x-1)) + (2*orig_gs->at<uint8_t>(y-1,x)) + (orig_gs->at<uint8_t>(y-1,x+1)) +
            (-1*orig_gs->at<uint8_t>(y+1,x-1)) + (-2*orig_gs->at<uint8_t>(y+1,x)) + (-1*orig_gs->at<uint8_t>(y+1,x+1));
            edges_cpu->at<uint8_t>(y,x) = sqrt((dx*dx)+(dy*dy));
            
        }
    }

}

void sobel_omp(const cv::Mat* orig_gs, cv::Mat* edges_omp, const unsigned int width, const unsigned int height) {
    omp_set_num_threads(4);
    
    #pragma omp parallel for
    for(int y = 1; y < height-1; y++) {
        for(int x = 1; x < width-1; x++) {
            int dx = (-1* (orig_gs->at<uint8_t>(y-1,x-1))) + (-2*(orig_gs->at<uint8_t>(y,x-1))) + (-1*(orig_gs->at<uint8_t>(y+1,x-1))) +
            (orig_gs->at<uint8_t>(y-1,x+1)) + (2*(orig_gs->at<uint8_t>(y,x+1))) + ((orig_gs->at<uint8_t>(y+1,x+1)));
            int dy = (orig_gs->at<uint8_t>(y-1,x-1)) + (2*orig_gs->at<uint8_t>(y-1,x)) + (orig_gs->at<uint8_t>(y-1,x+1)) +
            (-1*orig_gs->at<uint8_t>(y+1,x-1)) + (-2*orig_gs->at<uint8_t>(y+1,x)) + (-1*orig_gs->at<uint8_t>(y+1,x+1));
            edges_omp->at<uint8_t>(y,x) = sqrt((dx*dx)+(dy*dy));
            //printf("Thread %3d is running on cpu %3d\n", omp_get_thread_num(), sched_getcpu());
        }
    }
}

int avg(int* fpsMeanVec){
    int sum = 0;
    for(int i=0; i<meanLength; i++)
    {
        sum = sum + fpsMeanVec[i];
    }
    return (int)sum/meanLength;
}